#include "hip/hip_runtime.h"

__global__ void kernel_acc( float* posx, float * posy, float * posz,float *velx, float *vely, float * velz, float * accelx, float * accely, float * accelz,  float * masse ) {
	int h = 0, M = 10;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int k = i;
	int size2 =1024;
	float3 acc;
	__shared__ float m[1024],px[1024],py[1024],pz[1024] ;

	//for ( j = 0; j < size2; j++) {
if( i < size2){
	//float px = posx[i];
	//float py = posy[i];
	//float pz = posz[i];
	m[i]  = masse[i];
	px[i] = posx[i];
	py[i] = posy[i];
	pz[i] = posz[i];
	acc.x = 0.0;
	acc.y = 0.0;
	acc.z = 0.0;
		//accelx [i]  =0;
		//accely [i]  =0;	
		//accelz [i]  =0;
	//}
			//int ax = 0;
			//int ay = 0;
			//int az = 0;
	//for(k=0; k<size2; k++){
	    for(h=0;h<size2 ; h++){
		//if ( h != i ) {
			float mulPosX = px[h]-px[i];
			float mulPosY = py[h]-py[i];			
			float mulPosZ = pz[h]-pz[i];
			float d = sqrtf(mulPosX*mulPosX + mulPosY*mulPosY + mulPosZ*mulPosZ);
			if ( d < 1.0 ) d = 1.0;
			float l = m[h]* M * (1/(d*d*d));
			acc.x += mulPosX*l;
			acc.y += mulPosY*l;	
			acc.z += mulPosZ*l;

		//}
            }

	//accelx[i]= accx; 
	//accely[i]= accy; 
	//accelz[i]= accz;

	//posx[i]= 0.0;

	velx[i]+=acc.x;
	vely[i]+=acc.y;
	velz[i]+=acc.z;

        posx[i]+=velx[i]*0.1;
        posy[i]+=vely[i]*0.1;
        posz[i]+=velz[i]*0.1;

}


/*
	int k = i;

	if( k< size2){
			velx[k]+=accelx [k];
			vely[k]+=accely [k];
			velz[k]+=accelz [k];
		        posx[k]+=velx[k]*0.1;
		        posy[k]+=vely[k]*0.1;
		        posz[k]+=velz[k]*0.1;
	}
*/

}



/*
__global__ void kernel_pos(float* posx, float * posy, float * posz,float* velx, float * vely, float * velz, float * accelx, float * accely, float * accelz ) {


int size2 =1024;
//		for(k=0; k<size2; k=k+2){
	int k = blockIdx.x * blockDim.x + threadIdx.x;

	if( k< size2){

			velx[k]+=accelx [k];
			vely[k]+=accely [k];
			velz[k]+=accelz [k];
		        posx[k]+=velx[k]*0.1;
		        posy[k]+=vely[k]*0.1;
		        posz[k]+=velz[k]*0.1;

int l = 0;
	}


		//}
}
*/


void saxpy( int nblocks, int nthreads,float* posx, float * posy, float * posz,float* velx, float * vely, float * velz, float * accelx, float * accely, float * accelz,  float * masse ) {
	kernel_acc<<<nblocks, nthreads>>>(  posx, posy, posz,velx, vely,  velz, accelx, accely, accelz,   masse );
	//kernel_pos<<<nblocks, nthreads>>>(  posx,  posy, posz, velx, vely,  velz,  accelx, accely, accelz );
}
